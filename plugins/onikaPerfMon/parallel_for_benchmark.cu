#include "hip/hip_runtime.h"
/*
Licensed to the Apache Software Foundation (ASF) under one
or more contributor license agreements.  See the NOTICE file
distributed with this work for additional information
regarding copyright ownership.  The ASF licenses this file
to you under the Apache License, Version 2.0 (the
"License"); you may not use this file except in compliance
with the License.  You may obtain a copy of the License at

  http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied.  See the License for the
specific language governing permissions and limitations
under the License.
*/

#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

namespace onika { namespace scg
{

  struct IterativeBenchmarkFunctor
  {
    double * const __restrict__ m_array = nullptr;
    const long m_iterations = 0;
  
    ONIKA_HOST_DEVICE_FUNC inline void operator () ( size_t idx ) const
    {
      double x = m_array[idx];
      double y = pow(x,sin(x));
      for(long i=0;i<m_iterations;i++)
      {
        x = x + y;
        y = pow(x,sin(x));
      }
      m_array[idx] = x * y;
    }
  };

} }


namespace onika { namespace parallel
{
  template<> struct ParallelForFunctorTraits< onika::scg::IterativeBenchmarkFunctor >
  {      
    static inline constexpr bool CudaCompatible = true;
  };
}}

namespace onika { namespace scg
{

  class ParallelForBenchmark : public OperatorNode
  {
    using DoubleArray = onika::memory::CudaMMVector<double>;
  
    ADD_SLOT( long        , samples    , INPUT , 4096 , DocString{"Number of terms to compute"} );
    ADD_SLOT( long        , iterations , INPUT , 4096 , DocString{"Number of iterations for each term"} );
    ADD_SLOT( long        , block_size , INPUT ,  256 , DocString{"Thread teams (aka Cuda block) size"} );
    ADD_SLOT( DoubleArray , scratch    , PRIVATE );

  public:

    inline void execute () override final
    {
      scratch->resize( *samples );
      IterativeBenchmarkFunctor benchmark = { scratch->data() , *iterations };
      parallel_for( *samples , benchmark , parallel_execution_context() );
    }
  };
  
  // === register factories ===  
  ONIKA_AUTORUN_INIT(parallel_for_benchmark)
  {
   OperatorNodeFactory::instance()->register_factory( "parallel_for_benchmark", make_compatible_operator< ParallelForBenchmark > );
  }

} }


