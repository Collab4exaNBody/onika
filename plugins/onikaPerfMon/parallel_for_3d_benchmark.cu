#include "hip/hip_runtime.h"
/*
Licensed to the Apache Software Foundation (ASF) under one
or more contributor license agreements.  See the NOTICE file
distributed with this work for additional information
regarding copyright ownership.  The ASF licenses this file
to you under the Apache License, Version 2.0 (the
"License"); you may not use this file except in compliance
with the License.  You may obtain a copy of the License at

  http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied.  See the License for the
specific language governing permissions and limitations
under the License.
*/

#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/block_parallel_for.h>
#include <onika/parallel/parallel_for.h>

namespace onika { namespace scg
{

  struct GridBlock3DBenchmarkFunctor
  {
    double * const __restrict__ m_array = nullptr;
    const long m_size = 0;
  
    ONIKA_HOST_DEVICE_FUNC inline void operator () ( onikaInt3_t coord ) const
    {
      //ONIKA_CU_SHARED sum;
      const ssize_t idx = ONIKA_CU_BLOCK_IDX;
      ONIKA_CU_ATOMIC_ADD( m_array[idx] , 1.0 );
    }
  };

  struct Grid3DBenchmarkFunctor
  {
    double * const __restrict__ m_array = nullptr;
    const long m_size = 0;
  
    ONIKA_HOST_DEVICE_FUNC inline void operator () ( onikaInt3_t coord ) const
    {
      //printf("KERNEL %d,%d,%d\n",int(coord.x),int(coord.y),int(coord.z));
      const unsigned long N = m_size;
      const unsigned long i = coord.x;
      const unsigned long j = coord.y;
      const unsigned long k = coord.z;
      const ssize_t idx = (k*N+j)*N+i;
      ONIKA_CU_ATOMIC_ADD( m_array[idx] , 1.0 );
    }
  };

} }


namespace onika { namespace parallel
{
  template<> struct BlockParallelForFunctorTraits< onika::scg::Grid3DBenchmarkFunctor >
  {      
    static inline constexpr bool CudaCompatible = true;
  };
}}

namespace onika { namespace scg
{

  class ParallelFor3DBenchmark : public OperatorNode
  {
    using DoubleArray = onika::memory::CudaMMVector<double>;
  
    ADD_SLOT( long        , pfor3d_block_side , INPUT , 4 , DocString{"Thread teams (aka Cuda block) size"} );
    ADD_SLOT( long        , pfor3d_side , INPUT , 16 , DocString{"Number of terms to compute"} );
    ADD_SLOT( DoubleArray , scratch     , PRIVATE );

  public:

    inline void execute () override final
    {
      using onika::parallel::ParallelExecutionSpace;
      using onika::parallel::block_parallel_for;
      using onika::parallel::parallel_for;
      
      lout << "block_parallel_for 3D test" << std::endl;
      {
        const ssize_t N = *pfor3d_block_side;
        scratch->assign( N * N * N , 0.0 );
        Grid3DBenchmarkFunctor benchmark = { scratch->data() , N };
        ParallelExecutionSpace<3> parallel_range = { {0,0,0} , {N,N,N} };
        block_parallel_for( parallel_range , benchmark , parallel_execution_context() );
        for(int k=0;k<N;k++)
        {
          lout<<"K = "<<k<<std::endl;
          for(int j=0;j<N;j++)
          {
            lout<<"  J = "<<j<<std::endl<<"   ";
            for(int i=0;i<N;i++)
            {
              lout <<" "<< i<<":"<<scratch->at( (k*N+j)*N+i );
            }
            lout << std::endl;
          }
        }
      }

      lout << "parallel_for 3D test" << std::endl;
      {
        const ssize_t N = *pfor3d_side;
        scratch->assign( N * N * N , 0.0 );
        Grid3DBenchmarkFunctor benchmark = { scratch->data() , N };
        ParallelExecutionSpace<3> parallel_range = { {2,3,3} , {N-1,N-3,N-3} };
        
        parallel_for( parallel_range , benchmark , parallel_execution_context() );

        for(int k=0;k<N;k++)
        {
          lout<<"K="<<k<<std::endl;
          for(int j=0;j<N;j++)
          {
            lout<<"  J="<<std::setw(3)<<j<<" :";
            for(int i=0;i<N;i++)
            {
              lout <<" "<<int( scratch->at( (k*N+j)*N+i ) );
            }
            lout << std::endl;
          }
        }

      }
    }
  };
  
  // === register factories ===  
  ONIKA_AUTORUN_INIT(parallel_for_benchmark)
  {
   OperatorNodeFactory::instance()->register_factory( "parallel_for_3d_benchmark", make_compatible_operator< ParallelFor3DBenchmark > );
  }

} }


