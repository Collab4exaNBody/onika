#include "hip/hip_runtime.h"
/*
Licensed to the Apache Software Foundation (ASF) under one
or more contributor license agreements.  See the NOTICE file
distributed with this work for additional information
regarding copyright ownership.  The ASF licenses this file
to you under the Apache License, Version 2.0 (the
"License"); you may not use this file except in compliance
with the License.  You may obtain a copy of the License at

  http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied.  See the License for the
specific language governing permissions and limitations
under the License.
*/

#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/block_parallel_for.h>

namespace onika { namespace scg
{

  struct Grid3DBenchmarkFunctor
  {
    double * const __restrict__ m_array = nullptr;
    const long m_size = 0;
  
    ONIKA_HOST_DEVICE_FUNC inline void operator () ( onikaInt3_t coord ) const
    {
      //ONIKA_CU_SHARED sum;
      const ssize_t idx = ONIKA_CU_BLOCK_IDX;
      const double x = m_array[idx];
      const double y = x*x - 2*x + 1;
      m_array[idx] = x + y;
    }
  };




} }


namespace onika { namespace parallel
{
  template<> struct BlockParallelForFunctorTraits< onika::scg::Grid3DBenchmarkFunctor >
  {      
    static inline constexpr bool CudaCompatible = true;
  };
}}

namespace onika { namespace scg
{

  class ParallelFor3DBenchmark : public OperatorNode
  {
    using DoubleArray = onika::memory::CudaMMVector<double>;
  
    ADD_SLOT( long        , grid_size  , INPUT , 256 , DocString{"Number of terms to compute"} );
    ADD_SLOT( long        , block_size , INPUT , 256 , DocString{"Thread teams (aka Cuda block) size"} );
    ADD_SLOT( DoubleArray , scratch    , PRIVATE );

  public:

    inline void execute () override final
    {
      using onika::parallel::ParallelExecutionSpace;
    
      const ssize_t N = *grid_size;
      scratch->resize( N * N * N , 0.0 );
      Grid3DBenchmarkFunctor benchmark = { scratch->data() , N };
      ParallelExecutionSpace<3> grid = { {0,0,0} , {N,N,N} };
      block_parallel_for( grid , benchmark , parallel_execution_context() );
    }
  };
  
  // === register factories ===  
  ONIKA_AUTORUN_INIT(parallel_for_benchmark)
  {
   OperatorNodeFactory::instance()->register_factory( "parallel_for_3d_benchmark", make_compatible_operator< ParallelFor3DBenchmark > );
  }

} }


