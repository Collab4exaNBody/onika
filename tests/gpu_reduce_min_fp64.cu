#include "hip/hip_runtime.h"

// sur hfgpu
// nvcc -arch=compute_80 -code=sm_80 --compiler-options -fopenmp benchmark.cu -o benchmark_hfgpu
// execavec memoire unifiee
// OMP_NUM_THREADS=64 ccc_mprun -n1 -c128 -phfgpu -T3600 ./benchmark_hfgpu <<< "0 1 1037"
// avec memoire device et copie host/device
// OMP_NUM_THREADS=64 ccc_mprun -n1 -c128 -phfgpu -T3600 ./benchmark_hfgpu <<< "0 0 1037"

// sur HE
// nvcc -arch=compute_80 -code=sm_80 --compiler-options -fopenmp benchmark.cu -o benchmark_he
// nvcc -arch=compute_90 -code=sm_90 --compiler-options -fopenmp benchmark.cu -o benchmark_he
// execavec memoire unifiee
// OMP_NUM_THREADS=64 ./benchmark_he <<< "0 1 1037"
// avec memoire device et copie host/device
// OMP_NUM_THREADS=64 ./benchmark_he <<< "0 0 1037"

#include <iostream>
#include <chrono>
#include <omp.h>

//#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/cuda/cuda_context.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_execution_context.h>

#define N (1024*1024)

void cpu_init( double * __restrict__ data )
{
# pragma omp parallel
  {
#   pragma omp single
    {
      std::cout << "using "<<omp_get_num_threads()<<" CPU threads"<<std::endl;
    }
#   pragma omp for schedule(static)
    for(int i = 0; i < N; i++)
    {
      data[i] = i * 1.0 / N;
    }
  }
}

double cpu_compute( double * __restrict__ data )
{
  double data_min = std::numeric_limits<double>::max();
# pragma omp parallel for schedule(static) reduction(min:data_min)
  for(int i = 0; i < N; i++)
  {
    if( data[i] < data_min ) data_min = data[i];
  }
  return data_min;
}

int main()
{
  int n_gpus = 0;
  onika::cuda::CudaContext::set_global_gpu_enable( true );
  auto cu_dev_count_rc = ONIKA_CU_GET_DEVICE_COUNT(&n_gpus);
  if( n_gpus > 0 )
  {
    ONIKA_CU_CHECK_ERRORS( cu_dev_count_rc );
  }
  else
  {
    std::cout<<"no GPU found, aborting" << std::endl;
    return 1;
  }
  std::cout << "found "<<n_gpus<<" GPU(s)"<<std::endl;
  onika::memory::GenericHostAllocator::set_cuda_enabled( true );
  onika::parallel::ParallelExecutionContext::s_gpu_sm_mult    = 2;
  onika::parallel::ParallelExecutionContext::s_gpu_sm_add     = 0;
  onika::parallel::ParallelExecutionContext::s_gpu_block_size = 256;

  double *h_data = nullptr;
  double *d_data = nullptr;

  int run_host=0, uvm=0, idx=0;
  std::cin >> run_host >> uvm >> idx;
  std::cout << "run_host="<<run_host<<", uvm="<<uvm<<" , idx="<<idx<<std::endl;

  if( uvm )
  {
    ONIKA_CU_MALLOC_MANAGED( & h_data , N * sizeof(double) );
    d_data = h_data;
  }
  else
  {
    h_data = new double[N];
  }

  cpu_init( h_data );

  if( ! uvm )
  {
    ONIKA_CU_MALLOC( & d_data, N * sizeof(double));
    ONIKA_CU_MEMCPY( d_data, h_data, N * sizeof(double) /*, onikaMemcpyHostToDevice */ );
  }

  const auto T0 = std::chrono::high_resolution_clock::now();

  double vhost = 0.0;
  if(run_host) vhost = cpu_compute( h_data );

  const auto T1 = std::chrono::high_resolution_clock::now();

  // TODO: GPU reduction
  std::cout << "GPU reduction not implemented yet" << std::endl;
  const auto T2 = std::chrono::high_resolution_clock::now();

  if( ! uvm ) ONIKA_CU_MEMCPY( h_data, d_data, sizeof(double) /*, onikaMemcpyDeviceToHost */ );
  ONIKA_CU_DEVICE_SYNCHRONIZE();
  const double vcuda = h_data[0];
  const auto T3 = std::chrono::high_resolution_clock::now();

  std::cout << "result = "<< vhost<<" / "<<vcuda<<std::endl;
  if(run_host) std::cout << "host time = "<< (T1-T0).count() / 1000000.0 << std::endl;
  std::cout << "cuda time = "<< (T2-T1).count() / 1000000.0 << " + "<< (T3-T2).count() / 1000000.0 << " = "<< (T3-T1).count() / 1000000.0 <<std::endl;
  if(run_host) std::cout << "ratio = "<< (T1-T0).count() * 1.0 / (T3-T1).count()  << std::endl;

  if( ! uvm ) ONIKA_CU_FREE(d_data);

  return 0;
}

