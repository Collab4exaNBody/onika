
// sur hfgpu
// nvcc -arch=compute_80 -code=sm_80 --compiler-options -fopenmp benchmark.cu -o benchmark_hfgpu
// execavec memoire unifiee
// OMP_NUM_THREADS=64 ccc_mprun -n1 -c128 -phfgpu -T3600 ./benchmark_hfgpu <<< "0 1 1037"
// avec memoire device et copie host/device
// OMP_NUM_THREADS=64 ccc_mprun -n1 -c128 -phfgpu -T3600 ./benchmark_hfgpu <<< "0 0 1037"

// sur HE
// nvcc -arch=compute_80 -code=sm_80 --compiler-options -fopenmp benchmark.cu -o benchmark_he
// nvcc -arch=compute_90 -code=sm_90 --compiler-options -fopenmp benchmark.cu -o benchmark_he
// execavec memoire unifiee
// OMP_NUM_THREADS=64 ./benchmark_he <<< "0 1 1037"
// avec memoire device et copie host/device
// OMP_NUM_THREADS=64 ./benchmark_he <<< "0 0 1037"

#include <iostream>
#include <chrono>
#include <omp.h>

//#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/cuda/cuda_context.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_execution_context.h>

#define N (1024*1024)
#define M (10000000)

void cpu_init( double * __restrict__ data )
{
# pragma omp parallel
  {
#   pragma omp single
    {
      std::cout << "using "<<omp_get_num_threads()<<" CPU threads"<<std::endl;
    }
#   pragma omp for schedule(static)
    for(int i = 0; i < N; i++)
    {
      data[i] = i * 1.0 / N;
    }
  }
}

void cpu_compute( double * __restrict__ data )
{
# pragma omp parallel for schedule(static)
  for(int i = 0; i < N; i++)
  {
    for(int j = 0; j < M; j++)
    {
       data[i] = data[i] * data[i] - 0.25;
    }
  }
}

ONIKA_DEVICE_KERNEL_FUNC void gpu_compute( double * __restrict__ data )
{
  const int i = ONIKA_CU_THREAD_IDX + ONIKA_CU_BLOCK_IDX * ONIKA_CU_BLOCK_SIZE;
  for(int j = 0; j < M; j++)
  {
    data[i] = data[i] * data[i] - 0.25;
  }
}

int main()
{
  int n_gpus = 0;
  onika::cuda::CudaContext::set_global_gpu_enable( true );
  auto cu_dev_count_rc = ONIKA_CU_GET_DEVICE_COUNT(&n_gpus);
  if( n_gpus > 0 )
  {
    ONIKA_CU_CHECK_ERRORS( cu_dev_count_rc );
  }
  else
  {
    std::cout<<"no GPU found, aborting" << std::endl;
    return 1;
  }
  std::cout << "found "<<n_gpus<<" GPU(s)"<<std::endl;
  onika::memory::GenericHostAllocator::set_cuda_enabled( true );
  onika::parallel::ParallelExecutionContext::s_gpu_sm_mult    = 2;
  onika::parallel::ParallelExecutionContext::s_gpu_sm_add     = 0;
  onika::parallel::ParallelExecutionContext::s_gpu_block_size = 256;

  double *h_data = nullptr;
  double *d_data = nullptr;

  int run_host=0, uvm=0, idx=0;
  std::cin >> run_host >> uvm >> idx;
  std::cout << "run_host="<<run_host<<", uvm="<<uvm<<" , idx="<<idx<<std::endl;

  if( uvm )
  {
    ONIKA_CU_MALLOC_MANAGED( & h_data , N * sizeof(double) );
    d_data = h_data;
  }
  else
  {
    h_data = new double[N];
  }

  cpu_init( h_data );

  if( ! uvm )
  {
    ONIKA_CU_MALLOC( & d_data, N * sizeof(double));
    ONIKA_CU_MEMCPY( d_data, h_data, N * sizeof(double) /*, onikaMemcpyHostToDevice */ );
  }

  const auto T0 = std::chrono::high_resolution_clock::now();

  if(run_host) cpu_compute( h_data );
  const double vhost = h_data[idx];

  const auto T1 = std::chrono::high_resolution_clock::now();
  ONIKA_CU_LAUNCH_KERNEL(N/256,256,0,0,gpu_compute,d_data);
  const auto T2 = std::chrono::high_resolution_clock::now();

  if( ! uvm ) ONIKA_CU_MEMCPY( h_data, d_data, N * sizeof(double) /*, onikaMemcpyDeviceToHost */ );
  ONIKA_CU_DEVICE_SYNCHRONIZE();
  const double vcuda = h_data[idx];
  const auto T3 = std::chrono::high_resolution_clock::now();

  std::cout << "result["<<idx<<"] = "<< vhost<<" / "<<vcuda<<std::endl;
  if(run_host) std::cout << "host time = "<< (T1-T0).count() / 1000000.0 << std::endl;
  std::cout << "cuda time = "<< (T2-T1).count() / 1000000.0 << " + "<< (T3-T2).count() / 1000000.0 << " = "<< (T3-T1).count() / 1000000.0 <<std::endl;
  if(run_host) std::cout << "ratio = "<< (T1-T0).count() * 1.0 / (T3-T1).count()  << std::endl;

  if( ! uvm ) ONIKA_CU_FREE(d_data);

  return 0;
}

